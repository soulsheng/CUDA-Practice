#include "hip/hip_runtime.h"

#include "rgb2gray_cpu.h"

#include <hip/hip_runtime.h>

#include "defineMacro.h"


__global__ void rgb2gray_kernel3( ColorRGB* rgb, float* gray, int size )
{
	int index = blockIdx.x*blockDim.x + threadIdx.x;

	if(  index > size )
		return;
	ColorRGB color = rgb[index];
	gray[ index ] = 
		color.x * R_RATIO 
	+	color.y * G_RATIO 
	+	color.z * B_RATIO  ;
}

void rgb2gray_gpu3( float* rgb, float* gray, int size )
{
	float* d_arrayA ;
	hipMalloc( (void**)&d_arrayA, sizeof(float)*size*3 );
	hipMemcpy( d_arrayA, rgb, sizeof(float)*size*3, hipMemcpyHostToDevice );

	float* d_arrayC ;
	hipMalloc( (void**)&d_arrayC, sizeof(float)*size );

	int sizeBlock = size>BLOCKDIM?BLOCKDIM: size;
	int countBlock = (size+ sizeBlock-1)/sizeBlock;
	rgb2gray_kernel3<<< countBlock, sizeBlock >>>(  (ColorRGB*)d_arrayA,  d_arrayC, size );

	hipMemcpy( gray, d_arrayC, sizeof(float)*size, hipMemcpyDeviceToHost );

	hipFree( d_arrayA );
	hipFree( d_arrayC );

}

__global__ void rgb2gray_kernel2( float* rgb, float* gray, int size )
{
	int index = blockIdx.x*blockDim.x + threadIdx.x;

	if(  index > size )
		return;
	
	gray[ index ] = 
		rgb[ index +0] * R_RATIO 
	+	rgb[ index +size] * G_RATIO 
	+	rgb[ index +size*2] * B_RATIO  ;
}

void rgb2gray_gpu2( float* rgb, float* gray, int size )
{
	float* d_arrayA ;
	hipMalloc( (void**)&d_arrayA, sizeof(float)*size*3 );
	hipMemcpy( d_arrayA, rgb, sizeof(float)*size*3, hipMemcpyHostToDevice );

	float* d_arrayC ;
	hipMalloc( (void**)&d_arrayC, sizeof(float)*size );

	int sizeBlock = size>BLOCKDIM?BLOCKDIM: size;
	int countBlock = (size+ sizeBlock-1)/sizeBlock;
	rgb2gray_kernel2<<< countBlock, sizeBlock >>>(  d_arrayA,  d_arrayC, size );

	hipMemcpy( gray, d_arrayC, sizeof(float)*size, hipMemcpyDeviceToHost );

	hipFree( d_arrayA );
	hipFree( d_arrayC );

}

__global__ void rgb2gray_kernel1( float* rgb, float* gray, int size )
{
	int index = blockIdx.x*blockDim.x + threadIdx.x;

	if(  index > size )
		return;
	
	gray[ index ] =
		rgb[ index*3 ] * R_RATIO 
	+	rgb[ index*3 +1] * G_RATIO 
	+	rgb[ index*3 +2] * B_RATIO  ;
}

void rgb2gray_gpu1( float* rgb, float* gray, int size )
{
	float* d_arrayA ;
	hipMalloc( (void**)&d_arrayA, sizeof(float)*size*3 );
	hipMemcpy( d_arrayA, rgb, sizeof(float)*size*3, hipMemcpyHostToDevice );

	float* d_arrayC ;
	hipMalloc( (void**)&d_arrayC, sizeof(float)*size );

	int sizeBlock = size>BLOCKDIM?BLOCKDIM: size;
	int countBlock = (size+ sizeBlock-1)/sizeBlock;
	rgb2gray_kernel1<<< countBlock, sizeBlock >>>( d_arrayA, d_arrayC, size );

	hipMemcpy( gray, d_arrayC, sizeof(float)*size, hipMemcpyDeviceToHost );

	hipFree( d_arrayA );
	hipFree( d_arrayC );

}

void warnup_gpu( float* rgb, float* gray, int size )
{
	rgb2gray_gpu1( rgb, gray, size );
}
