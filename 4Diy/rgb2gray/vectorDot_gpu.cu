#include "hip/hip_runtime.h"

#include "vectorDot_cpu.h"

#include <hip/hip_runtime.h>

#define  BLOCKDIM	256


__global__ void vectorDot_kernel2( float* arrayA, float* arrayB, float* arrayC, int size )
{
	int bid = blockIdx.x;
	int tid = threadIdx.x;
	int index = bid*blockDim.x + tid;

	if(  index > size )
		return;
	
	arrayC[ index ] =    arrayA[ index +0] * arrayB[ index +0] 
	+ arrayA[ index +size] * arrayB[ index +size] 
	+ arrayA[ index +size*2] * arrayB[ index +size*2]  ;
}

void vectorDot_gpu2( float* arrayA, float* arrayB, float* arrayC, int size )
{
	float* d_arrayA ;
	hipMalloc( (void**)&d_arrayA, sizeof(float)*size*3 );

	hipMemcpy( d_arrayA, arrayA, sizeof(float)*size*3, hipMemcpyHostToDevice );

	float* d_arrayB ;
	hipMalloc( (void**)&d_arrayB, sizeof(float)*size*3 );

	hipMemcpy( d_arrayB, arrayB, sizeof(float)*size*3, hipMemcpyHostToDevice );

	float* d_arrayC ;
	hipMalloc( (void**)&d_arrayC, sizeof(float)*size );

	int sizeBlock = size>BLOCKDIM?BLOCKDIM: size;
	int countBlock = (size+ sizeBlock-1)/sizeBlock;
	vectorDot_kernel2<<< countBlock, sizeBlock >>>(  d_arrayA,  d_arrayB, d_arrayC, size );

	hipMemcpy( arrayC, d_arrayC, sizeof(float)*size, hipMemcpyDeviceToHost );

	hipFree( d_arrayA );
	hipFree( d_arrayB );
	hipFree( d_arrayC );

}

__global__ void vectorDot_kernel1( float* arrayA, float* arrayB, float* arrayC, int size )
{
	int bid = blockIdx.x;
	int tid = threadIdx.x;
	int index = bid*blockDim.x + tid;

	if(  index > size )
		return;
	
	arrayC[ index ] = 
		arrayA[ index*3 ] * arrayB[ index*3 ] 
	+ arrayA[ index*3 +1] * arrayB[ index*3 +1] 
	+ arrayA[ index*3 +2] * arrayB[ index*3 +2]  ;
}

void vectorDot_gpu1( float* arrayA, float* arrayB, float* arrayC, int size )
{
	float* d_arrayA ;
	hipMalloc( (void**)&d_arrayA, sizeof(float)*size*3 );

	hipMemcpy( d_arrayA, arrayA, sizeof(float)*size*3, hipMemcpyHostToDevice );

	float* d_arrayB ;
	hipMalloc( (void**)&d_arrayB, sizeof(float)*size*3 );

	hipMemcpy( d_arrayB, arrayB, sizeof(float)*size*3, hipMemcpyHostToDevice );

	float* d_arrayC ;
	hipMalloc( (void**)&d_arrayC, sizeof(float)*size );

	int sizeBlock = size>BLOCKDIM?BLOCKDIM: size;
	int countBlock = (size+ sizeBlock-1)/sizeBlock;
	vectorDot_kernel1<<< countBlock, sizeBlock >>>( d_arrayA, d_arrayB, d_arrayC, size );

	hipMemcpy( arrayC, d_arrayC, sizeof(float)*size, hipMemcpyDeviceToHost );

	hipFree( d_arrayA );
	hipFree( d_arrayB );
	hipFree( d_arrayC );

}

void warnup_gpu( float* arrayA, float* arrayB, float* arrayC, int size )
{
	vectorDot_gpu1( arrayA, arrayB, arrayC, size );
}
