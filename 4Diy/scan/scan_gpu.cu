#include "hip/hip_runtime.h"

#include "scan_cpu.h"

#include <hip/hip_runtime.h>

#define  BLOCKDIM	(1<<9)

__global__ void scan_kernel1( float* array, int size )
{
	int row = threadIdx.x;

	int WIDTH = size/BLOCKDIM ;

	for (int i=1;i<WIDTH;i++)
	{
		array[i + row*WIDTH] += array[i-1 + row*WIDTH];
	}

}

float scan_gpu1( float* array, int size )
{
	float* d_array ;
	hipMalloc( (void**)&d_array, sizeof(float)*size );

	hipMemcpy( d_array, array, sizeof(float)*size, hipMemcpyHostToDevice );

	int sizeBlock = size/BLOCKDIM > BLOCKDIM?BLOCKDIM: size/BLOCKDIM;
	int countBlock = 1;
	scan_kernel1<<< 1, sizeBlock >>>( d_array, size );

	hipMemcpy( array, d_array, sizeof(float)*size, hipMemcpyDeviceToHost );

	hipFree( d_array );

	return array[size-1];
}

__global__ void scan_kernel2( float* array, int size )
{
	int index = blockIdx.x * blockDim.x + threadIdx.x;

	if( index > size )
		return;

	__shared__ float sdata[BLOCKDIM*2];

	sdata[threadIdx.x] = array[index];
	__syncthreads();

	int first = 0;

	for ( int d=1;d<=blockDim.x/2; d+=d, first=BLOCKDIM-first )
	{
		if( threadIdx.x < d )
			sdata[threadIdx.x+BLOCKDIM-first] = sdata[threadIdx.x+first];
		else
			sdata[threadIdx.x+BLOCKDIM-first] = sdata[threadIdx.x+first] + sdata[threadIdx.x-d+first];
		__syncthreads();
	}

	array[index] = sdata[threadIdx.x+first];
	__syncthreads();
}

float scan_gpu2( float* array, int size )
{
	float* d_array ;
	hipMalloc( (void**)&d_array, sizeof(float)*size );

	hipMemcpy( d_array, array, sizeof(float)*size, hipMemcpyHostToDevice );

	int sizeBlock = size>BLOCKDIM?BLOCKDIM: size;
	int countBlock = (size+ sizeBlock-1)/sizeBlock;
	scan_kernel2<<< countBlock, sizeBlock >>>( d_array, size );

	hipMemcpy( array, d_array, sizeof(float)*size, hipMemcpyDeviceToHost );

	hipFree( d_array );

	return array[size-1];
}

void warnup_gpu( float* array, int size )
{
	scan_gpu2( array, size );
}
