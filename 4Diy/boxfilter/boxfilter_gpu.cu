#include "hip/hip_runtime.h"

#include "boxfilter_cpu.h"

#include <hip/hip_runtime.h>

#define  BLOCKDIM_MAX	(1<<10)

__global__ void boxfilter_kernel1( float* array, int size, int width )
{
	for (int i=1;i<width;i++)
	{
		array[i + threadIdx.x*width] += array[i-1 + threadIdx.x*width];
	}

}

float boxfilter_gpu1( float* array, int size, int width )
{
	float* d_array ;
	hipMalloc( (void**)&d_array, sizeof(float)*size );

	hipMemcpy( d_array, array, sizeof(float)*size, hipMemcpyHostToDevice );

	int sizeBlock = size/width > BLOCKDIM_MAX?BLOCKDIM_MAX: size/width;
	int countBlock = 1;
	boxfilter_kernel1<<< 1, sizeBlock >>>( d_array, size, width );

	hipMemcpy( array, d_array, sizeof(float)*size, hipMemcpyDeviceToHost );

	hipFree( d_array );

	return array[size-1];
}

__global__ void boxfilter_kernel2( float* array, int size, int width )
{
	int index = blockIdx.x * width + threadIdx.x;

	if( index > size )
		return;

	__shared__ float sdata[BLOCKDIM_MAX*2];

	sdata[threadIdx.x] = array[index];
	__syncthreads();

	int first = 0;

	for ( int d=1;d<=blockDim.x/2; d+=d, first=blockDim.x-first )
	{
		if( threadIdx.x < d )
			sdata[threadIdx.x+blockDim.x-first] = sdata[threadIdx.x+first];
		else
			sdata[threadIdx.x+blockDim.x-first] = sdata[threadIdx.x+first] + sdata[threadIdx.x-d+first];
		__syncthreads();
	}

	array[index] = sdata[threadIdx.x+first];
	__syncthreads();
}

float boxfilter_gpu2( float* array, int size, int width )
{
	float* d_array ;
	hipMalloc( (void**)&d_array, sizeof(float)*size );

	hipMemcpy( d_array, array, sizeof(float)*size, hipMemcpyHostToDevice );

	int sizeBlock = width>BLOCKDIM_MAX?BLOCKDIM_MAX: width;
	int countBlock = size / width ;// ������һ��block����һ��
	boxfilter_kernel2<<< countBlock, sizeBlock >>>( d_array, size, width );

	hipMemcpy( array, d_array, sizeof(float)*size, hipMemcpyDeviceToHost );

	hipFree( d_array );

	return array[size-1];
}

void warnup_gpu( float* array, int size, int width )
{
	boxfilter_gpu2( array, size, width );
}
