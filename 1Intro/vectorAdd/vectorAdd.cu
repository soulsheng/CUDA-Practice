/**
 * Copyright 1993-2013 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */

/**
 * Vector addition: C = A + B.
 *
 * This sample is a very basic sample that implements element by element
 * vector addition. It is the same as the sample illustrating Chapter 2
 * of the programming guide with some additions like error checking.
 */

#include <stdio.h>

// For the CUDA runtime routines (prefixed with "cuda_")
#include <hip/hip_runtime.h>

/**
 * CUDA Kernel Device code
 *
 * Computes the vector addition of A and B into C. The 3 vectors have the same
 * number of elements numElements.
 */
__global__ void
vectorAdd(const float *A, const float *B, float *C, int numElements)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < numElements)
    {
        C[i] = A[i] + B[i];
    }
}

/**
 * Host main routine
 */
int
main(void)
{

    // 第1步，设置向量维度
    int numElements = 50000;
    size_t size = numElements * sizeof(float);
    printf("[Vector addition of %d elements]\n", numElements);

    // 第2步，为向量分配内存
	float *h_A, *h_B, *h_C;
	h_A = h_B = h_C = NULL;
    h_A = (float *)malloc(size);
    h_B = (float *)malloc(size);
    h_C = (float *)malloc(size);

    // 第3步，内存向量赋值，随机向量
    for (int i = 0; i < numElements; ++i)
    {
        h_A[i] = rand()/(float)RAND_MAX;
        h_B[i] = rand()/(float)RAND_MAX;
    }

    // 第4步，为向量分配显存
    float *d_A, *d_B, *d_C;
	d_A = d_B = d_C = NULL;
	hipMalloc((void **)&d_A, size);
	hipMalloc((void **)&d_B, size);
	hipMalloc((void **)&d_C, size);

    // 第5步，显存向量赋值，从内存拷贝
    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
	hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    // 第6步，设置线程维度
    int threadsPerBlock = 256;
    int blocksPerGrid =(numElements + threadsPerBlock - 1) / threadsPerBlock;

	// 第7步，调用GPU执行计算
    vectorAdd<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, numElements);

    // 第8步，收获成果，从显存拷回内存
    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);


    // 第9步，验证成果是否正确
    for (int i = 0; i < numElements; ++i)
    {
        if (fabs(h_A[i] + h_B[i] - h_C[i]) > 1e-5)
        {
            fprintf(stderr, "Result verification failed at element %d!\n", i);
            exit(EXIT_FAILURE);
        }
    }
    printf("Test PASSED\n");

    // 第10步，释放显存和内存
    hipFree(d_A);
	hipFree(d_B);
	hipFree(d_C);

    free(h_A);
    free(h_B);
    free(h_C);

    // 第11步，重置CUDA，完成
    hipDeviceReset();

	// 第12步，完成任务，退出
    printf("Done\n");
    return 0;
}

