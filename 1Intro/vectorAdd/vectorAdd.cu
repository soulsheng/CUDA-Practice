/**
 * Copyright 1993-2013 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */

/**
 * Vector addition: C = A + B.
 *
 * This sample is a very basic sample that implements element by element
 * vector addition. It is the same as the sample illustrating Chapter 2
 * of the programming guide with some additions like error checking.
 */

#include <stdio.h>

// For the CUDA runtime routines (prefixed with "cuda_")
#include <hip/hip_runtime.h>

/**
 * CUDA Kernel Device code
 *
 * Computes the vector addition of A and B into C. The 3 vectors have the same
 * number of elements numElements.
 */
__global__ void
vectorAdd(const float *A, const float *B, float *C, int numElements)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < numElements)
    {
        C[i] = A[i] + B[i];
    }
}

/**
 * Host main routine
 */
int
main(void)
{

    // ��1������������ά��
    int numElements = 50000;
    size_t size = numElements * sizeof(float);
    printf("[Vector addition of %d elements]\n", numElements);

    // ��2����Ϊ���������ڴ�
	float *h_A, *h_B, *h_C;
	h_A = h_B = h_C = NULL;
    h_A = (float *)malloc(size);
    h_B = (float *)malloc(size);
    h_C = (float *)malloc(size);

    // ��3�����ڴ�������ֵ���������
    for (int i = 0; i < numElements; ++i)
    {
        h_A[i] = rand()/(float)RAND_MAX;
        h_B[i] = rand()/(float)RAND_MAX;
    }

    // ��4����Ϊ���������Դ�
    float *d_A, *d_B, *d_C;
	d_A = d_B = d_C = NULL;
	hipMalloc((void **)&d_A, size);
	hipMalloc((void **)&d_B, size);
	hipMalloc((void **)&d_C, size);

    // ��5�����Դ�������ֵ�����ڴ濽��
    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
	hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    // ��6���������߳�ά��
    int threadsPerBlock = 256;
    int blocksPerGrid =(numElements + threadsPerBlock - 1) / threadsPerBlock;

	// ��7��������GPUִ�м���
    vectorAdd<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, numElements);

    // ��8�����ջ�ɹ������Դ濽���ڴ�
    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);


    // ��9������֤�ɹ��Ƿ���ȷ
    for (int i = 0; i < numElements; ++i)
    {
        if (fabs(h_A[i] + h_B[i] - h_C[i]) > 1e-5)
        {
            fprintf(stderr, "Result verification failed at element %d!\n", i);
            exit(EXIT_FAILURE);
        }
    }
    printf("Test PASSED\n");

    // ��10�����ͷ��Դ���ڴ�
    hipFree(d_A);
	hipFree(d_B);
	hipFree(d_C);

    free(h_A);
    free(h_B);
    free(h_C);

    // ��11��������CUDA�����
    hipDeviceReset();

	// ��12������������˳�
    printf("Done\n");
    return 0;
}

