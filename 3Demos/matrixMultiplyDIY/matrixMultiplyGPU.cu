#include "hip/hip_runtime.h"

#include <hip/hip_runtime.h>

#include <iostream>
using namespace std;

#include "matrixMultiplyGPU.cuh"
#include "timerCUDA.h"

#define  TILE 16

// GPU �汾1����ʼ
__global__ void kernelMatrixMul1( float* a, float*b, float*c, int n )
{
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	int x = blockIdx.x * blockDim.x + threadIdx.x;

	if( x >= n || y >=n )
		return;

	for(int k=0;k<n;k++)
		c[ y*n +x] += a[ y*n + k] * b[ k*n +x] ;
}

// GPU �汾2��block�ֿ飬DIY
__global__ void kernelMatrixMul2( float* a, float*b, float*c, int n )
{
	
	int blockIdy = blockIdx.y ;

	int blockIdxx = blockIdx.x ;
	
	int threadIdy = threadIdx.y;
	
	int threadIdxx = threadIdx.x;

	float cBlockOne = 0.0f;

	for(int k=0;k<n/TILE;k++)
	{
		// ��һ��������С��ռ�
		__shared__ float aBlock[TILE][TILE];
		__shared__ float bBlock[TILE][TILE];

		// �ڶ�������ȡС���ڴ���еĴ洢λ��
		int aOffset =( blockIdy * n/TILE) * (TILE*TILE) + k*TILE  ;
		int bOffset =( k * n/TILE) * (TILE*TILE) + blockIdxx*TILE  ;

		// ��������С�鸳ֵ
		{
			aBlock[threadIdy][threadIdxx] = a[ aOffset + threadIdy*n + threadIdxx ];
			bBlock[threadIdy][threadIdxx] = b[ bOffset + threadIdy*n + threadIdxx ];
		}
		__syncthreads();

		// ���Ĳ���С�������		
		for(int p=0;p<TILE;p++)
		{
			cBlockOne += aBlock[threadIdy][p] * bBlock[p][threadIdxx];
		}
			
				
	}

	int cOffset =( blockIdy * n/TILE) * (TILE*TILE) + blockIdxx*TILE  ;
	// ���岽��С������˽���ۼӵ������		
	c[ cOffset + threadIdy*n + threadIdxx] = cBlockOne;

}

// GPU �汾3��block�ֿ飬SDK
template <int BLOCK_SIZE> 
__global__ void kernelMatrixMul3( float *A, float *B, float *C, int wA, int wB )
{
	
    // Block index
	int bx = blockIdx.x ;
	int by = blockIdx.y ;
	
    // Thread index
	int tx = threadIdx.x;
	int ty = threadIdx.y;


    // Index of the first sub-matrix of A processed by the block
	int aBegin = wA * BLOCK_SIZE * by ;

    // Index of the last sub-matrix of A processed by the block
	int aEnd   = aBegin + wA - 1;

    // Step size used to iterate through the sub-matrices of A
	int aStep  = BLOCK_SIZE;

    // Index of the first sub-matrix of B processed by the block
	int bBegin = BLOCK_SIZE * bx;

    // Step size used to iterate through the sub-matrices of B
	int bStep  = BLOCK_SIZE * wB;

	// Csub is used to store the element of the block sub-matrix
    // that is computed by the thread
	float Csub = 0.0f;

	for(int a = aBegin, b = bBegin; 
		a <= aEnd; 
		a += aStep, b += bStep)
	{
		// ��һ��������С��ռ�
		__shared__ float As[BLOCK_SIZE][BLOCK_SIZE];
		__shared__ float Bs[BLOCK_SIZE][BLOCK_SIZE];

		// ��������С�鸳ֵ
		As[ty][tx] = A[ a + wA * ty+ tx ];
		Bs[ty][tx] = B[ b + wB * ty+ tx ];
		
		__syncthreads();

		// ���Ĳ���С�������		
#pragma unroll
		for(int k = 0; k < BLOCK_SIZE; k++ )
		{
			Csub += As[ty][k] * Bs[k][tx];
		}
		
		// Synchronize to make sure that the preceding
        // computation is done before loading two new
        // sub-matrices of A and B in the next iteration
        __syncthreads();
	}

	int c = wB * BLOCK_SIZE * by + BLOCK_SIZE * bx;
	// ���岽��С������˽���ۼӵ������		
	C[ c + wB * ty + tx] = Csub;

}

// GPU �汾1����ʼ
void matrixMulGPU1( float* a, float*b, float*c, int n, bool bTimeKernel )
{
	float *aDev,*bDev,*cDev;
	hipMalloc( (void**)&aDev, n*n*sizeof(float) );
	hipMalloc( (void**)&bDev, n*n*sizeof(float) );
	hipMalloc( (void**)&cDev, n*n*sizeof(float) );

	hipMemcpy( aDev, a, n*n*sizeof(float), hipMemcpyHostToDevice );
	hipMemcpy( bDev, b, n*n*sizeof(float), hipMemcpyHostToDevice );
	hipMemset( cDev, 0, n*n*sizeof(float) );

	int nBlock = 16;
	int nGrid = (n + nBlock-1)/nBlock;
	dim3 sizeBlock(nBlock, nBlock);
	dim3 sizeGrid( nGrid, nGrid );
	
	timerCUDA	timerGPU;
	if( bTimeKernel )
		timerGPU.start();

	kernelMatrixMul1<<< sizeGrid,sizeBlock >>>( aDev, bDev, cDev, n );
	hipError_t err = hipGetLastError();

	if( err != hipSuccess )
		cout << "error" << endl;
	
	if( bTimeKernel )
	{
		timerGPU.stop();
		cout << "Kernel time : " << timerGPU.getTime() << endl;
	}

	hipMemcpy( c, cDev, n*n*sizeof(float), hipMemcpyDeviceToHost );

	hipFree( aDev );
	hipFree( bDev );
	hipFree( cDev );
}

// GPU �汾2��block�ֿ�
void matrixMulGPU2( float* a, float*b, float*c, int n, bool bTimeKernel )
{
	float *aDev,*bDev,*cDev;
	hipMalloc( (void**)&aDev, n*n*sizeof(float) );
	hipMalloc( (void**)&bDev, n*n*sizeof(float) );
	hipMalloc( (void**)&cDev, n*n*sizeof(float) );

	hipMemcpy( aDev, a, n*n*sizeof(float), hipMemcpyHostToDevice );
	hipMemcpy( bDev, b, n*n*sizeof(float), hipMemcpyHostToDevice );
	hipMemset( cDev, 0, n*n*sizeof(float) );

	int nBlock = 16;
	int nGrid = (n + nBlock-1)/nBlock;
	dim3 sizeBlock(nBlock, nBlock);
	dim3 sizeGrid( nGrid, nGrid );

	timerCUDA	timerGPU;
	if( bTimeKernel )
		timerGPU.start();

	kernelMatrixMul2<<< sizeGrid,sizeBlock >>>( aDev, bDev, cDev, n );
	hipError_t err = hipGetLastError();

	if( err != hipSuccess )
		cout << "error" << endl;

	if( bTimeKernel )
	{
		timerGPU.stop();
		cout << "Kernel time : " << timerGPU.getTime() << endl;
	}

	hipMemcpy( c, cDev, n*n*sizeof(float), hipMemcpyDeviceToHost );

	hipFree( aDev );
	hipFree( bDev );
	hipFree( cDev );
}

// GPU �汾3��block�ֿ飬SDK
void matrixMulGPU3( float* a, float*b, float*c, int n, bool bTimeKernel )
{
	float *aDev,*bDev,*cDev;
	hipMalloc( (void**)&aDev, n*n*sizeof(float) );
	hipMalloc( (void**)&bDev, n*n*sizeof(float) );
	hipMalloc( (void**)&cDev, n*n*sizeof(float) );

	hipMemcpy( aDev, a, n*n*sizeof(float), hipMemcpyHostToDevice );
	hipMemcpy( bDev, b, n*n*sizeof(float), hipMemcpyHostToDevice );
	hipMemset( cDev, 0, n*n*sizeof(float) );

	int nBlock = 16;
	int nGrid = (n + nBlock-1)/nBlock;
	dim3 sizeBlock(nBlock, nBlock);
	dim3 sizeGrid( nGrid, nGrid );

	timerCUDA	timerGPU;
	if( bTimeKernel )
		timerGPU.start();

	kernelMatrixMul3<16><<< sizeGrid,sizeBlock >>>( aDev, bDev, cDev, n, n );
	hipError_t err = hipGetLastError();

	if( err != hipSuccess )
		cout << "error" << endl;

	if( bTimeKernel )
	{
		timerGPU.stop();
		cout << "Kernel time : " << timerGPU.getTime() << endl;
	}

	hipMemcpy( c, cDev, n*n*sizeof(float), hipMemcpyDeviceToHost );

	hipFree( aDev );
	hipFree( bDev );
	hipFree( cDev );
}

void setupCUDA()
{
	hipSetDevice( 0 );
}