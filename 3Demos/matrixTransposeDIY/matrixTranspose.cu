#include "hip/hip_runtime.h"
#include "matrixTranspose.h"



__global__ void transposeSimple(float *odata, float *idata, int size_x, int size_y)
{
	//2D thread ID	
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;

	int index_in = row*size_x + col;
	int index_out = col*size_x + row;

	odata[index_out] =  idata[index_in];
}

__global__ void transposeSharedBlock(float *odata, float *idata, int size_x, int size_y)
{
    __shared__ float tile[BLOCK_SIZE][BLOCK_SIZE+1]; // +1 ���bank confict

    int xIndex = blockIdx.x * BLOCK_SIZE + threadIdx.x;
    int yIndex = blockIdx.y * BLOCK_SIZE + threadIdx.y;
    int index_in = xIndex + (yIndex)*size_x;

    xIndex = blockIdx.y * BLOCK_SIZE + threadIdx.x;
    yIndex = blockIdx.x * BLOCK_SIZE + threadIdx.y;
    int index_out = xIndex + (yIndex)*size_y;

    for (int i=0; i<BLOCK_SIZE; i+=BLOCK_SIZE)
    {
      tile[threadIdx.y+i][threadIdx.x] = idata[index_in+i*size_x];
    }
    
    __syncthreads();

    for (int i=0; i<BLOCK_SIZE; i+=BLOCK_SIZE)
    {
      odata[index_out+i*size_y] = tile[threadIdx.x][threadIdx.y+i];
    }
}

extern "C"
void MatrixTransposeOnDevice(float *P, float *M, unsigned int size_x, unsigned int size_y, bool bTimerKernel/*=false*/)
{
	int size= size_x*size_y*sizeof(float);

	//Interface host call to the device kernel code and invoke the kernel

	float *M_d=NULL;
    hipMalloc((void**)&M_d,size);
	hipMemcpy(M_d, M,size,hipMemcpyHostToDevice);

	float *P_d=NULL;
	hipMalloc((void**)&P_d,size);
	hipMemcpy(P_d, P,size,hipMemcpyHostToDevice);

	//kernel invocation code
	dim3 dimBlock(BLOCK_SIZE,BLOCK_SIZE);
	dim3 dimGrid(size_x/BLOCK_SIZE,size_y/BLOCK_SIZE);


	hipEvent_t start, stop;

	if( bTimerKernel )
	{
		hipEventCreate(&start);
		hipEventCreate(&stop);

		// Record the start event
		hipEventRecord(start, NULL);
	}

#if 0
	//Matrix Transpose Kernel Simple
	transposeSimple<<<dimGrid,dimBlock>>>(P_d,M_d,size_x,size_y);
#else
	//Matrix Transpose Kernel Shared
	transposeSharedBlock<<<dimGrid,dimBlock>>>(P_d,M_d,size_x,size_y);
#endif

	if( bTimerKernel )
	{
		// Record the stop event
		hipEventRecord(stop, NULL);

		// Wait for the stop event to complete
		hipEventSynchronize(stop);

		float msecTotal = 0.0f;
		hipEventElapsedTime(&msecTotal, start, stop);

		printf(	"GPU Time Kernel = %.3f msec \n",msecTotal );
	}

	//read P from the device
	hipMemcpy(P,P_d,size,hipMemcpyDeviceToHost);

	//free device matrices
	hipFree(M_d);
	hipFree(P_d);

}