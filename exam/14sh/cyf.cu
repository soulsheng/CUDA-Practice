#include "hip/hip_runtime.h"
#include <stdio.h>
#include <assert.h>
#include <stdlib.h>   
#include <malloc.h>

#include <math.h>
#include <conio.h>
#include <hip/hip_runtime.h>

#include <memory.h> 
//
#define N 8192      //�ܼ����ģ
#define S_DIM 512    //����ģ�ķֿ�
#define B_DIM 32    //�������ֵ�ķֿ�

#define MAXMIN_B_DIM 256
#define MAXMIN_DIM 1024  //�������Сֵ�ķֿ�
bool verify(float *ab,float *abGpu,int n)
{
	for(int i=0;i<n;i++){
		if(ab[i]!=0 && fabs(ab[i]-abGpu[i])/fabs(ab[i])>1e-5){
			return false;
		}
	}
	return true;
}

//-----------------------cpu�����Ӻ���---------------------------------
float   cpuAmpR(int *R)
{
   float sum=0.0;
   for(unsigned int  i=0;i<N;i++)
  {
	  float Rtmp=R[i];
      sum+=Rtmp*Rtmp;
   }
   return sqrtf(sum);
}


void cpuCountA(int *R,float Amp,float *A)
{
	for(int i=0;i<N;i++){
		for(int j=0;j<N;j++){
			int idx=i*N+j;
			float Ri=R[i];
			float Rj=R[j];
			A[idx]=Ri*Rj/Amp;
		}
	}
}
float cpuMaxA(float *A)
{
	float MaxA=A[0];
	for(int i=0;i<N;i++){
		for(int j=0;j<N;j++){
			int idx=i*N+j;
			if(A[idx]>MaxA){
				MaxA=A[idx];
			}
		}
	}
	return MaxA;
}
float cpuMinA(float *A)
{
	float MinA=A[0];
	for(int i=0;i<N;i++){
		for(int j=0;j<N;j++){
			int idx=i*N+j;
			if(A[idx]<MinA){
				MinA=A[idx];
			}
		}
	}
	return MinA;
}
//=============================================================

//-------------------------GPU�˺���---------------------------

__global__   void  gpuAmpR(int *R,float *SumTemp)
{
	__shared__ float SData[S_DIM];
	int  bx=blockIdx.x;  
	int  tx=threadIdx.x;
	float Rtmp=R[bx*blockDim.x+tx];
	SData[tx]=Rtmp*Rtmp;

	__syncthreads();
	for(unsigned int s=blockDim.x/2;s>0;s>>=1){
		if(tx<s){
			SData[tx]+=SData[tx+s];
		}
		__syncthreads();
	}

	if(tx==0) SumTemp[bx]=SData[0];

}
__global__ void gpuCountA(int *R,float *Amp,float *A)
{
	int  bx=blockIdx.x;  
	int  by=blockIdx.y;
	int  tx=threadIdx.x;
	int  ty=threadIdx.y;
	int Idx;

	__shared__  float AS[B_DIM];
	__shared__  float BS[B_DIM];
	if(ty==0){
		if(bx*B_DIM+tx<N)   
			AS[tx] = R[bx*B_DIM+tx];  
		else  
			AS[tx] = 0;
	}
	if(ty==1){
		if(by*B_DIM+tx<N)   
			BS[tx] = R[by*B_DIM+tx];  
		else  
			BS[tx] = 0;
	}
	__syncthreads();
	Idx=(by*B_DIM+ty)*N+bx*B_DIM+tx;
	if(Idx<N*N){
		A[Idx]=AS[tx]*BS[ty]/(*Amp);
	}
}
__global__ void gpuMaxA(float *A,float *MaxTemp)
{
	__shared__ float SData[MAXMIN_DIM];
	__shared__ float max_Block;
	int  bx=blockIdx.x;  
	int  tx=threadIdx.x;
	

	int tid=bx*blockDim.x+tx;
	int Times=(N*N+gridDim.x*blockDim.x-1)/(gridDim.x*blockDim.x);
	for(int i=0;i<Times;i++){
		if(tid<N*N){
			SData[tx]=A[tid];
		}
		else{
			SData[tx]=0;
		}
		__syncthreads();
		for(unsigned int s=blockDim.x/2;s>0;s>>=1){
			if(tx<s){
				if(SData[tx]<SData[tx+s]){
					SData[tx+s]=SData[tx];
				}
			}
			__syncthreads();
		}
		if(tx==0){
			if(i==0){
				max_Block=SData[0];
			}
			else{
				if(max_Block<SData[0]){
					max_Block=SData[0];
				}
			}
		}
		__syncthreads();
		tid+=gridDim.x*blockDim.x;
	}
	if(tx==0){
		MaxTemp[bx]=max_Block;
	}
}

__global__ void gpuMinA(float *A,float *MinTemp)
{
	__shared__ float SData[MAXMIN_DIM];
	__shared__ float min_Block;
	int  bx=blockIdx.x;  
	int  tx=threadIdx.x;
	

	int tid=bx*blockDim.x+tx;
	int Times=(N*N+gridDim.x*blockDim.x-1)/(gridDim.x*blockDim.x);
	for(int i=0;i<Times;i++){
		if(tid<N*N){
			SData[tx]=A[tid];
		}
		else{
			SData[tx]=0;
		}
		__syncthreads();
		for(unsigned int s=blockDim.x/2;s>0;s>>=1){
			if(tx<s){
				if(SData[tx]>SData[tx+s]){
					SData[tx+s]=SData[tx];
				}
			}
			__syncthreads();
		}
		if(tx==0){
			if(i==0){
				min_Block=SData[0];
			}
			else{
				if(min_Block<SData[0]){
					min_Block=SData[0];
				}
			}
		}
		__syncthreads();
		tid+=gridDim.x*blockDim.x;
	}
	if(tx==0){
		MinTemp[bx]=min_Block;
	}
}



//============================================================


void main()
{
	int *data_R;
	float *SumR,SumR2,temp;
	float *Amp_R,*A_array,*A_min,*A_max,*MaxTemp,*MinTemp;//,*data_b,*data_c1,*data_c2;
	bool verifySeccuss=0;
	int blockNum=(N+S_DIM-1)/S_DIM;
	hipMallocManaged(&data_R,N*sizeof(int));
	hipMallocManaged(&Amp_R,sizeof(float));
	hipMallocManaged(&SumR,blockNum*sizeof(float));
	hipMallocManaged(&A_max,sizeof(float));
	hipMallocManaged(&A_min,sizeof(float));
	hipMallocManaged(&MaxTemp,MAXMIN_B_DIM*sizeof(float));
	hipMallocManaged(&MinTemp,MAXMIN_B_DIM*sizeof(float));
	hipMallocManaged(&A_array,N*N*sizeof(float));

	for(int i=0;i<N;i++){
		data_R[i]=rand();
	}

	*Amp_R=cpuAmpR(data_R);
	cpuCountA(data_R,*Amp_R,A_array);
	*A_max=cpuMaxA(A_array);
	*A_min=cpuMinA(A_array);



	printf("CPU ��������\n|R| %f�����ֵ %f����Сֵ %f\n",*Amp_R,*A_max,*A_min);


	gpuAmpR<<<blockNum,S_DIM>>>(data_R,SumR);
	hipDeviceSynchronize();
	SumR2=0.0;
	for (int i=0;i<blockNum;i++){
		SumR2+=SumR[i];
	}

	*Amp_R=sqrt(SumR2);
	dim3 mygrid(((N+B_DIM-1)/B_DIM),(N+B_DIM-1)/B_DIM);  
	dim3 myblock(B_DIM,B_DIM);  

	gpuCountA<<<mygrid,myblock>>>(data_R,Amp_R,A_array);


	gpuMaxA<<<MAXMIN_B_DIM,MAXMIN_DIM>>>(A_array,MaxTemp);
	gpuMinA<<<MAXMIN_B_DIM,MAXMIN_DIM>>>(A_array,MinTemp);

	hipDeviceSynchronize();
	temp=MaxTemp[0];
	for(int i=1;i<MAXMIN_B_DIM;i++){
		if(temp<MaxTemp[i]){
			temp=MaxTemp[i];
		}
	}
	*A_max=temp;

	temp=MinTemp[0];
	for(int i=1;i<MAXMIN_B_DIM;i++){
		if(temp>MinTemp[i]){
			temp=MinTemp[i];
		}
	}
	*A_min=temp;

	*A_max=cpuMaxA(A_array);
	*A_min=cpuMinA(A_array);

	printf("GPU ��������\n|R| %f�����ֵ %f����Сֵ %f\n",*Amp_R,*A_max,*A_min);
	//verifySeccuss=verify(data_c1,data_c2,1);

	//if(verifySeccuss)
	//	printf("Verify Seccuss.\n");
	//else
	//	printf("Verify Error!\n");
	hipFree(data_R);
	hipFree(Amp_R);
	hipFree(SumR);
	hipFree(A_max);
	hipFree(A_min);
	hipFree(MaxTemp);
	hipFree(MinTemp);
	
	getch();
}