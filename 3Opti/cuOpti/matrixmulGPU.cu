#include "hip/hip_runtime.h"

#include <hip/hip_runtime.h>

#include <iostream>
using namespace std;

#include "matrixmulGPU.cuh"

__global__ void kernelMatrixMul1( float* a, float*b, float*c, int n )
{
	int ii = blockIdx.x * blockDim.x + threadIdx.x;

	if( ii >= n*n )
		return;

	int i= ii / n ;
	int j= ii %n;

	for(int k=0;k<n;k++)
		c[ i*n +j] += a[ i*n + k] * b[ k*n +j] ;
}

__global__ void kernelMatrixMul2( float* a, float*b, float*c, int n )
{
}

__global__ void kernelMatrixMul3( float* a, float*b, float*c, int n )
{
}

void matrixMulGPU1( float* a, float*b, float*c, int n )
{
	float *aDev,*bDev,*cDev;
	hipMalloc( (void**)&aDev, n*n*sizeof(float) );
	hipMalloc( (void**)&bDev, n*n*sizeof(float) );
	hipMalloc( (void**)&cDev, n*n*sizeof(float) );

	hipMemcpy( aDev, a, n*n*sizeof(float), hipMemcpyHostToDevice );
	hipMemcpy( bDev, b, n*n*sizeof(float), hipMemcpyHostToDevice );

	int nBlock = 256;
	int nGrid = (n*n + nBlock-1)/nBlock;

	kernelMatrixMul1<<< nGrid, nBlock >>>( aDev, bDev, cDev, n );
	hipError_t err = hipGetLastError();

	if( err != hipSuccess )
		cout << "error" << endl;

	hipMemcpy( c, cDev, n*n*sizeof(float), hipMemcpyDeviceToHost );

	hipFree( aDev );
	hipFree( bDev );
	hipFree( cDev );

}

void matrixMulGPU2( float* a, float*b, float*c, int n )
{
	float *aDev,*bDev,*cDev;
	hipMalloc( (void**)&aDev, n*n*sizeof(float) );
	hipMalloc( (void**)&bDev, n*n*sizeof(float) );
	hipMalloc( (void**)&cDev, n*n*sizeof(float) );

	hipMemcpy( aDev, a, n*n*sizeof(float), hipMemcpyHostToDevice );
	hipMemcpy( bDev, b, n*n*sizeof(float), hipMemcpyHostToDevice );

	int nBlock = 256;
	int nGrid = (n*n + nBlock-1)/nBlock;

	kernelMatrixMul2<<< nGrid, nBlock >>>( aDev, bDev, cDev, n );
	hipError_t err = hipGetLastError();

	if( err != hipSuccess )
		cout << "error" << endl;

	hipMemcpy( c, cDev, n*n*sizeof(float), hipMemcpyDeviceToHost );

	hipFree( aDev );
	hipFree( bDev );
	hipFree( cDev );
}

void matrixMulGPU3( float* a, float*b, float*c, int n )
{
}