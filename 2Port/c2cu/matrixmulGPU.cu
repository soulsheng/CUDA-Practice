
#include <hip/hip_runtime.h>

#include <iostream>
using namespace std;

__global__ void kernelMatrixMul( float* a, float*b, float*c, int n )
{
	int ii = blockIdx.x * blockDim.x + threadIdx.x;

	if( ii >= n*n )
		return;

	int i= ii / n ;
	int j= ii %n;

	for(int k=0;k<n;k++)
		c[ i*n +j] += a[ i*n + k] * b[ k*n +j] ;
}

extern "C" void matrixMulGPU( float* a, float*b, float*c, int n )
{
	float *aDev,*bDev,*cDev;
	hipMalloc( (void**)&aDev, n*n*sizeof(float) );
	hipMalloc( (void**)&bDev, n*n*sizeof(float) );
	hipMalloc( (void**)&cDev, n*n*sizeof(float) );

	hipMemcpy( aDev, a, n*n*sizeof(float), hipMemcpyHostToDevice );
	hipMemcpy( bDev, b, n*n*sizeof(float), hipMemcpyHostToDevice );

	int nBlock = 256;
	int nGrid = (n*n + nBlock-1)/nBlock;

	kernelMatrixMul<<< nGrid, nBlock >>>( aDev, bDev, cDev, n );
	hipError_t err = hipGetLastError();

	if( err != hipSuccess )
		cout << "error" << endl;

	hipMemcpy( c, cDev, n*n*sizeof(float), hipMemcpyDeviceToHost );
}